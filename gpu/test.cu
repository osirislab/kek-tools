#include "hip/hip_runtime.h"
#include "test.h"


__global__ void axpy(float a, float* x, float* y) {
  y[threadIdx.x] = a * x[threadIdx.x];
}


int test(const uint8_t *in_data, size_t len) {
  const int kDataLen = 4;

  float a = 2.0f;
  float host_x[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};
  float host_y[kDataLen];

  // Copy input data to device.
  float* device_x;
  float* device_y;
  hipMalloc(&device_x, kDataLen * sizeof(float));
  hipMalloc(&device_y, kDataLen * sizeof(float));
  hipMemcpy(device_x, host_x, kDataLen * sizeof(float),
             hipMemcpyHostToDevice);

  // Launch the kernel.
  axpy<<<1, kDataLen>>>(a, device_x, device_y);

  // Copy output data to host.
  hipDeviceSynchronize();
  hipMemcpy(host_y, device_y, kDataLen * sizeof(float),
             hipMemcpyDeviceToHost);

  // Print the results.
  for (int i = 0; i < kDataLen; ++i) {
    printf("y[%d] = %f\n", i, host_y[i]);
  }

  hipDeviceReset();
  return 0;

}

